#include "hip/hip_runtime.h"
// etape3_mpi_gpus : MPI + multi-GPU : résolution du problème de Jacobi avec plusieurs GPU sur un cluster.
// Solveur Jacobi sur grille 2D de taille N x N, T itérations.
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include <mpi.h>

#define N_GRILLE 4096
#define T 1000

__global__ void jacobi_kernel(double* A, double* B, int N_SIZE);

// Version MPI + multi-GPU
void jacobi_mpi_gpu(double* h_A, double* h_B, int N, int local_rows, int rank, int size, MPI_Comm comm) {
    double *d_A, *d_B;
    size_t local_size = (local_rows + 2) * N * sizeof(double); // +2 pour halos haut/bas

    hipSetDevice(rank % 8); // suppose max 8 GPUs par nœud

    hipMalloc(&d_A, local_size);
    hipMalloc(&d_B, local_size);

    // Copier la sous-grille locale (hors halos)
    hipMemcpy(d_A + N, h_A + N, local_rows * N * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_B + N, h_B + N, local_rows * N * sizeof(double), hipMemcpyHostToDevice);

    dim3 block(16, 16);
    dim3 grid((N-2+block.x-1)/block.x, (local_rows+block.y-1)/block.y);

    // Buffers pour halos sur le host
    double* halo_send_top = (double*)malloc(N * sizeof(double));
    double* halo_recv_top = (double*)malloc(N * sizeof(double));
    double* halo_send_bot = (double*)malloc(N * sizeof(double));
    double* halo_recv_bot = (double*)malloc(N * sizeof(double));

    for (int t = 0; t < T; t++) {
        // Copier les halos du device vers le host
        if (rank > 0) {
            hipMemcpy(halo_send_top, d_A + N, N * sizeof(double), hipMemcpyDeviceToHost);
        }
        if (rank < size-1) {
            hipMemcpy(halo_send_bot, d_A + local_rows*N, N * sizeof(double), hipMemcpyDeviceToHost);
        }

        // Échanges d’halos haut/bas
        // Envoyer/recevoir ligne du haut
        if (rank > 0) {
            MPI_Sendrecv(
                halo_send_top, N, MPI_DOUBLE, rank-1, 0,
                halo_recv_top, N, MPI_DOUBLE, rank-1, 1,
                comm, MPI_STATUS_IGNORE
            );
            // Copier la ligne reçue dans le halo du haut sur le device
            hipMemcpy(d_A, halo_recv_top, N * sizeof(double), hipMemcpyHostToDevice);
        }
        // Envoyer/recevoir ligne du bas
        if (rank < size-1) {
            MPI_Sendrecv(
                halo_send_bot, N, MPI_DOUBLE, rank+1, 1,
                halo_recv_bot, N, MPI_DOUBLE, rank+1, 0,
                comm, MPI_STATUS_IGNORE
            );
            // Copier la ligne reçue dans le halo du bas sur le device
            hipMemcpy(d_A + (local_rows+1)*N, halo_recv_bot, N * sizeof(double), hipMemcpyHostToDevice);
        }

        jacobi_kernel<<<grid, block>>>(d_A, d_B, N);
        hipDeviceSynchronize();
        double* tmp = d_A; d_A = d_B; d_B = tmp;
    }

    free(halo_send_top);
    free(halo_recv_top);
    free(halo_send_bot);
    free(halo_recv_bot);

    // Copier la sous-grille locale (hors halos) vers le host
    hipMemcpy(h_A + N, d_A + N, local_rows * N * sizeof(double), hipMemcpyDeviceToHost);

    hipFree(d_A);
    hipFree(d_B);
}

int main(int argc, char** argv) {
    MPI_Init(&argc, &argv);
    int rank, size;
    MPI_Comm comm = MPI_COMM_WORLD;
    MPI_Comm_rank(comm, &rank);
    MPI_Comm_size(comm, &size);

    if (N_GRILLE < 3 || T < 1) {
        if (rank == 0) {
            fprintf(stderr, "N_GRILLE must be >= 3 and T must be >= 1\n");
        }
        MPI_Finalize();
        return EXIT_FAILURE;
    }

    // We set the message only on rank 0
    if (rank == 0) {
        printf("Lancement de l'étape 3 : MPI + multi-GPU\n");
        printf("Grille de taille %d x %d, T = %d itérations\n", N_GRILLE, N_GRILLE, T);
    }

    int local_rows = (N_GRILLE-2) / size;
    int rem = (N_GRILLE-2) % size;
    if (rank < rem) local_rows++;

    // Allouer la sous-grille locale (+2 lignes pour halos)
    double *A = (double*)calloc((local_rows+2)*N_GRILLE, sizeof(double));
    double *B = (double*)calloc((local_rows+2)*N_GRILLE, sizeof(double));

    clock_t start = clock();
    jacobi_mpi_gpu(A, B, N_GRILLE, local_rows, rank, size, comm);
    clock_t end = clock();

    double local_time = ((double)(end - start)) / CLOCKS_PER_SEC;
    double max_time;
    MPI_Reduce(&local_time, &max_time, 1, MPI_DOUBLE, MPI_MAX, 0, comm);

    if (rank == 0) {
        printf("Terminé %s rank %d/%d\n", "etape3_mpi_gpus (MPI + multi-GPU)", rank, size);
        printf("Max GPU time: %.6f seconds rank %d/%d\n", max_time, rank, size);
    }

    free(A); free(B);
    MPI_Finalize();
    return 0;
}



#include <hip/hip_runtime.h>
__global__ void jacobi_kernel(double* A, double* B, int N_SIZE, int i_start, int i_end) {
    int i = blockIdx.y * blockDim.y + threadIdx.y + i_start;
    int j = blockIdx.x * blockDim.x + threadIdx.x + 1;
    if (i < i_end && j < N_SIZE-1) {
        B[i*N_SIZE + j] = 0.25 * (A[(i-1)*N_SIZE + j] + A[(i+1)*N_SIZE + j]
                            + A[i*N_SIZE + j-1] + A[i*N_SIZE + j+1]);
    }
}
